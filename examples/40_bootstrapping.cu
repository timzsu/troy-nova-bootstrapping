#include "hip/hip_runtime.h"
#include <random>

#include "examples.h"

using namespace std;
using namespace troy;
using namespace bootstrap;

void random_real(vector<double> &vec, size_t size) {
  random_device rn;
  mt19937_64 rnd(rn());
  thread_local std::uniform_real_distribution<double> distribution(-1, 1);

  vec.reserve(size);

  for (size_t i = 0; i < size; i++) {
    vec[i] = distribution(rnd);
  }
}

void example_bootstrapping() {
  long boundary_K = 25;
  long deg = 59;
  long scale_factor = 2;
  long inverse_deg = 1;

  // The following parameters have been adjusted to satisfy the memory constraints of an A100 GPU
  long logN = 15;  // 16 -> 15
  long loge = 10;

  long logn = 15;  // 14 -> 13
  long sparse_slots = (1 << logn);

  int logp = 46;
  int logq = 40;
  int log_special_prime = 51;

  int secret_key_hamming_weight = 0;

  int remaining_level = 3; 
  int boot_level = 14;  // >= subsum 1 + coefftoslot 2 + ModReduction 9 + slottocoeff 2
  int total_level = remaining_level + boot_level;

  vector<size_t> coeff_bit_vec;
  coeff_bit_vec.push_back(logq);
  for (int i = 0; i < remaining_level; i++) {
    coeff_bit_vec.push_back(logp);
  }
  for (int i = 0; i < boot_level; i++) {
    coeff_bit_vec.push_back(logq);
  }
  coeff_bit_vec.push_back(log_special_prime);

  std::cout << "Setting Parameters..." << endl;
  EncryptionParameters parms(SchemeType::CKKS);
  size_t poly_modulus_degree = (size_t)(1 << logN);
  double scale = pow(2.0, logp);

  parms.set_poly_modulus_degree(poly_modulus_degree);
  parms.set_coeff_modulus(CoeffModulus::create(poly_modulus_degree, coeff_bit_vec));
//   parms.set_secret_key_hamming_weight(secret_key_hamming_weight);
//   parms.set_sparse_slots(sparse_slots);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    print_parameters(*context);
    cout << endl;

    CKKSEncoder encoder(context);

  CKKSEvaluator ckks_evaluator(&parms, scale);

  size_t slot_count = ckks_evaluator.encoder.slot_count();

  // Bootstrapper bootstrapper(
  //     loge,
  //     logn,
  //     logN - 1,
  //     total_level,
  //     scale,
  //     boundary_K,
  //     deg,
  //     scale_factor,
  //     inverse_deg,
  //     &ckks_evaluator);

  // std::cout << "Generating Optimal Minimax Polynomials..." << endl;
  // bootstrapper.prepare_mod_polynomial();

  // std::cout << "Adding Bootstrapping Keys..." << endl;
  // vector<int> gal_steps_vector;
  // gal_steps_vector.push_back(0);
  // for (int i = 0; i < logN - 1; i++) {
  //   gal_steps_vector.push_back((1 << i));
  // }
  // bootstrapper.addLeftRotKeys_Linear_to_vector_3(gal_steps_vector);

  // ckks_evaluator.decryptor.create_galois_keys_from_steps(gal_steps_vector, *(ckks_evaluator.galois_keys));
  // std::cout << "Galois key generated from steps vector." << endl;

  // bootstrapper.slot_vec.push_back(logn);

  // std::cout << "Generating Linear Transformation Coefficients..." << endl;
  // bootstrapper.generate_LT_coefficient_3();

  vector<double> sparse(sparse_slots, 0.0);
  vector<double> input(slot_count, 0.0);
  vector<double> before(slot_count, 0.0);
  vector<double> after(slot_count, 0.0);

  random_real(sparse, sparse_slots);

  Plaintext plain;
  Ciphertext cipher;

  // Create input cipher
  for (size_t i = 0; i < slot_count; i++) {
    input[i] = sparse[i % sparse_slots];
  }

  ckks_evaluator.encoder.encode(input, scale, plain);
  ckks_evaluator.encoder.decode(plain, before);
  for (long i = 0; i < sparse_slots; i++) {
    if (i < 10) std::cout << input[i] << " <----> " << before[i] << endl;
  }

  ckks_evaluator.encryptor->encrypt_asymmetric(plain, cipher);

  // Mod switch to the lowest level
  for (int i = 0; i < total_level; i++) {
    ckks_evaluator.evaluator->mod_switch_to_next_inplace(cipher);
  }

  // Decrypt input cipher to obtain the original input
  ckks_evaluator.decryptor->decrypt(cipher, plain);
  ckks_evaluator.encoder.decode(plain, before);

  // auto start = system_clock::now();

  // Ciphertext rtn;
  // bootstrapper.bootstrap_3(rtn, cipher);

  // duration<double> sec = system_clock::now() - start;
  // std::cout << "Bootstrapping took: " << sec.count() << "s" << endl;
  // std::cout << "Return cipher level: " << rtn.coeff_modulus_size() << endl;

  // ckks_evaluator.decryptor.decrypt(rtn, plain);
  // ckks_evaluator.encoder.decode(plain, after);

  // double mean_err = 0;
  for (long i = 0; i < sparse_slots; i++) {
    if (i < 10) std::cout << input[i] << " <----> " << before[i] << endl;
    // mean_err += abs(before[i] - after[i]);
  }
  // mean_err /= sparse_slots;
  // std::cout << "Mean absolute error: " << mean_err << endl;
}
