#include "hip/hip_runtime.h"
#include <iostream>

#include "ckks_evaluator.cuh"
#include "utils.cuh"

namespace bootstrap {

void CKKSEvaluator::print_decoded_pt(troy::Plaintext &pt, int num) {
  vector<double> v;

  encoder.decode(pt, v);

  for (int i = 0; i < num; i++) {
    cout << v[i] << " ";
  }
  cout << endl;
}

void CKKSEvaluator::print_decrypted_ct(troy::Ciphertext &ct, int num) {
  troy::Plaintext temp;
  vector<double> v;

  // if (!ct.chain_index()) {
  //   cout << endl;
  //   return;
  // }

  decryptor->decrypt(ct, temp);
  encoder.decode(temp, v);

  for (int i = 0; i < num; i++) {
    cout << v[i] << " ";
  }
  cout << endl;
}

vector<double> CKKSEvaluator::init_vec_with_value(double value) {
  std::vector<double> vec(encoder.slot_count(), value);
  return vec;
}

troy::Plaintext CKKSEvaluator::init_plain_power_of_x(size_t exponent) {
  troy::Plaintext plain_power_of_x;

  vector<double> vec = init_vec_with_value(0.0);
  vec[exponent] = 1.0;

  encoder.encode(vec, scale, plain_power_of_x);
  return plain_power_of_x;
}

troy::Ciphertext CKKSEvaluator::init_guess(troy::Ciphertext x) {
  troy::Plaintext A, B;
  encoder.encode(-1.29054537e-04, scale, A);
  encoder.encode(1.29054537e-01, scale, B);
  return eval_line(x, A, B);
}

troy::Ciphertext CKKSEvaluator::eval_line(troy::Ciphertext x, troy::Plaintext m, troy::Plaintext c) {
  evaluator->mod_switch_plain_to_inplace(m, x.parms_id());
  evaluator->multiply_plain_inplace(x, m);
  evaluator->rescale_to_next_inplace(x);

  evaluator->mod_switch_plain_to_inplace(c, x.parms_id());
  x.scale() = scale;
  evaluator->add_plain_inplace(x, c);

  return x;
}

troy::Ciphertext CKKSEvaluator::invert_sqrt(troy::Ciphertext x, int d_newt, int d_gold) {
  troy::Ciphertext res = init_guess(x);
  troy::Ciphertext y = newton_iter(x, res, d_newt);
  pair<troy::Ciphertext, troy::Ciphertext> sqrt_inv_sqrt = goldschmidt_iter(x, y, d_gold);
  return sqrt_inv_sqrt.second;
}

uint64_t CKKSEvaluator::get_modulus(troy::Ciphertext &x, int k) {
  const troy::utils::ConstSlice<troy::Modulus>& modulus = context->get_context_data(x.parms_id())->get()->parms().coeff_modulus();
  int sz = modulus.size();
  return modulus[sz - k].value();
}

void CKKSEvaluator::re_encrypt(troy::Ciphertext &ct) {
  auto timer = bootstrap::Timer();
  while (ct.coeff_modulus_size() > 1) {
    evaluator->mod_switch_to_next_inplace(ct);
  }

  troy::Plaintext temp;
  vector<double> v;
  decryptor->decrypt(ct, temp);
  encoder.decode(temp, v);
  encoder.encode(v, ct.scale(), temp);
  encryptor->encrypt_asymmetric(temp, ct);

  timer.stop();
  cout << timer.duration() << " milliseconds" << endl;

  // cout << "Re-encrypted cipher depth: " <<
  // context.get_context_data(ct.parms_id()).chain_depth() << "\n";
}

pair<troy::Ciphertext, troy::Ciphertext> CKKSEvaluator::goldschmidt_iter(troy::Ciphertext v, troy::Ciphertext y, int d) {
  troy::Ciphertext x, h, r, temp;
  troy::Plaintext constant;

  encoder.encode(0.5, scale, constant);

  // GoldSchmidt's algorithm
  evaluator->mod_switch_to_inplace(v, y.parms_id());
  evaluator->multiply(v, y, x);
  evaluator->relinearize_inplace(x, relin_keys);
  evaluator->rescale_to_next_inplace(x);

  evaluator->mod_switch_plain_to_inplace(constant, y.parms_id());
  evaluator->multiply_plain(y, constant, h);
  evaluator->rescale_to_next_inplace(h);

  for (int i = 0; i < d; i++) {
    encoder.encode(0.5, scale, constant);

    evaluator->multiply(x, h, r);
    evaluator->relinearize_inplace(r, relin_keys);
    evaluator->rescale_to_next_inplace(r);

    r.scale() = scale;
    evaluator->negate(r, temp);
    evaluator->mod_switch_plain_to_inplace(constant, temp.parms_id());
    evaluator->add_plain(temp, constant, r);

    // x = x + x*r
    evaluator->mod_switch_to_inplace(x, r.parms_id());
    evaluator->multiply(x, r, temp);
    evaluator->relinearize_inplace(temp, relin_keys);
    evaluator->rescale_to_next_inplace(temp);

    x.scale() = scale;
    temp.scale() = scale;
    evaluator->mod_switch_to_inplace(x, temp.parms_id());
    evaluator->add_inplace(x, temp);

    // h = h + h*r
    evaluator->mod_switch_to_inplace(h, r.parms_id());
    evaluator->multiply(h, r, temp);
    evaluator->relinearize_inplace(temp, relin_keys);
    evaluator->rescale_to_next_inplace(temp);

    h.scale() = scale;
    temp.scale() = scale;
    evaluator->mod_switch_to_inplace(h, temp.parms_id());
    evaluator->add_inplace(h, temp);
  }

  encoder.encode(2.0, scale, constant);
  evaluator->mod_switch_plain_to_inplace(constant, h.parms_id());
  evaluator->multiply_plain_inplace(h, constant);
  evaluator->rescale_to_next_inplace(h);

  return make_pair(x, h);
}

troy::Ciphertext CKKSEvaluator::newton_iter(troy::Ciphertext x, troy::Ciphertext res, int iter) {
  for (int i = 0; i < iter; i++) {
    troy::Plaintext three_half, neg_half;

    encoder.encode(1.5, scale, three_half);
    encoder.encode(-0.5, scale, neg_half);

    // x^2
    troy::Ciphertext res_sq;
    evaluator->square(res, res_sq);
    evaluator->relinearize_inplace(res_sq, relin_keys);
    evaluator->rescale_to_next_inplace(res_sq);

    //-0.5*x*b
    troy::Ciphertext res_x;
    evaluator->mod_switch_plain_to_inplace(neg_half, x.parms_id());
    evaluator->multiply_plain(x, neg_half, res_x);
    evaluator->rescale_to_next_inplace(res_x);

    if (context->get_context_data(res.parms_id())->get()->chain_index() <
        context->get_context_data(res_x.parms_id())->get()->chain_index())
      evaluator->mod_switch_to_inplace(res_x, res.parms_id());
    else
      evaluator->mod_switch_to_inplace(res, res_x.parms_id());

    evaluator->multiply_inplace(res_x, res);
    evaluator->relinearize_inplace(res_x, relin_keys);
    evaluator->rescale_to_next_inplace(res_x);

    //-0.5*b*x^3
    evaluator->mod_switch_to_inplace(res_sq, res_x.parms_id());
    evaluator->multiply_inplace(res_x, res_sq);
    evaluator->relinearize_inplace(res_x, relin_keys);
    evaluator->rescale_to_next_inplace(res_x);

    // 1.5*x
    evaluator->mod_switch_plain_to_inplace(three_half, res.parms_id());
    evaluator->multiply_plain_inplace(res, three_half);
    evaluator->rescale_to_next_inplace(res);

    //-0.5*b*x^3 + 1.5*x
    evaluator->mod_switch_to_inplace(res, res_x.parms_id());
    res_x.scale() = scale;
    res.scale() = scale;
    evaluator->add_inplace(res, res_x);
  }

  return res;
}

void CKKSEvaluator::eval_odd_deg9_poly(vector<double> &a, troy::Ciphertext &x, troy::Ciphertext &dest) {
  /*
        (polyeval/odd9.h)
        P(x) = a9 x^9 + a7 x^7 + a5 x^5 + a3 x^3 + a1 x

        T1 = (a3 + a5 x^2) x^3
        T2 = (a7 x + a9 x^3) x^6
        T3 = a1 x
        P(x) = T1 + T2 + T3

        Depth=4, #Muls=5

        Exactly what babystep_giantstep would do, but written explicitly to optimize

        ###

        . Errorless Polynomial Evaluation (3.2. of https://eprint.iacr.org/2020/1203)
        GOAL: evaluate a polynomial exactly so no need to stabilize and lose precision
        (x at level L and scale D -. P(x) at level L-4 and scale D)
        it's possible to do this exactly for polyeval as (x,x2,x3,x6) determine the scale D_L for each involved level L:
        (assume the primes at levels L to L-4 are p, q, r, s)

        level       ctx       scale (D_l)
        ==================================
          L          x          D
          L-1        x2         D^2 / p
          L-2        x3         D^3 / pq
          L-3        x6         D^6 / p^2 q^2 r

        Just by encoding constants at different scales we can make every ctx at level l be at scale D_l
        (not possible in general, e.g. rescale(x2*x2) produces L-2 ciphertext with scale D^4/ppq)
        (to fix this we would use the Adjust op. that multiplies ctx by constants and Algo 3 for primes from https://eprint.iacr.org/2020/1118)

        Now we know that sc(P(x)) should be D, so we recursively go back to compute the scales for each coefficient
        sc(T1)=sc(T2)=sc(T3)=sc(P(x))=D

        T3:
            sc(a1) = q (should be p but it gets multiplied with modswitched x)

        T2:
            sc(x^6) = D^6 / p^2 q^2 r, so sc(a7*x) = sc(a9*x^3) = p^2 q^2 r s / D^5
            next, sc(a7) = p^2 q^3 r s / D^6
            similarly, sc(a9) = p^3 q^3 r^2 s / D^8

        T1:
            sc(x^3) = D^3 / pq
            implying sc(a3) = pqr / D^2 and also sc(a5*x^2) = pqr / D^2
            as sc(x^2) = D^2 / p this implies sc(a5) = p^2 q^2 r / D^4
    */
  // chrono::high_resolution_clock::time_point time_start, time_end;
  // time_start = high_resolution_clock::now();
  double D = x.scale();  // maybe not init_scale but preserved

  uint64_t p = get_modulus(x, 1);
  uint64_t q = get_modulus(x, 2);
  uint64_t r = get_modulus(x, 3);
  uint64_t s = get_modulus(x, 4);
  uint64_t t = get_modulus(x, 5);

  p = q;
  q = r;
  r = s;
  s = t;

  vector<double> a_scales(10);
  a_scales[1] = q;
  a_scales[3] = (double)p / D * q / D * r;
  a_scales[5] = (double)p / D * p / D * q / D * q / D * r;
  a_scales[7] = (double)p / D * p / D * q / D * q / D * q / D * r / D * s;
  a_scales[9] = (double)p / D * p / D * p / D * q / D * q / D * q / D * r / D * r / D * s;

  ///////////////////////////////////////////////
  troy::Ciphertext x2, x3, x6;

  evaluator->square(x, x2);
  evaluator->relinearize_inplace(x2, relin_keys);
  evaluator->rescale_to_next_inplace(x2);  // L-1

  evaluator->mod_switch_to_next_inplace(x);  // L-1
  evaluator->multiply(x2, x, x3);
  evaluator->relinearize_inplace(x3, relin_keys);
  evaluator->rescale_to_next_inplace(x3);  // L-2

  evaluator->square(x3, x6);
  evaluator->relinearize_inplace(x6, relin_keys);
  evaluator->rescale_to_next_inplace(x6);  // L-3

  troy::Plaintext a1, a3, a5, a7, a9;

  // Build T1
  troy::Ciphertext T1;
  double a5_scale = D / x2.scale() * p / x3.scale() * q;
  encoder.encode(a[5], x2.parms_id(), a5_scale, a5);  // L-1
  evaluator->multiply_plain(x2, a5, T1);
  evaluator->rescale_to_next_inplace(T1);  // L-2

  // Update: using a_scales[3] is only approx. correct, so we directly use T1.scale()
  encoder.encode(a[3], T1.parms_id(), T1.scale(), a3);  // L-2

  evaluator->add_plain_inplace(T1, a3);  // L-2
  evaluator->multiply_inplace(T1, x3);
  evaluator->relinearize_inplace(T1, relin_keys);
  evaluator->rescale_to_next_inplace(T1);  // L-3

  // Build T2
  troy::Ciphertext T2;
  troy::Plaintext a9_switched;
  double a9_scale = D / x3.scale() * r / x6.scale() * q;
  encoder.encode(a[9], x3.parms_id(), a9_scale, a9);  // L-2
  evaluator->multiply_plain(x3, a9, T2);
  evaluator->rescale_to_next_inplace(T2);  // L-3

  troy::Ciphertext a7x;
  double a7_scale = T2.scale() / x.scale() * p;
  encoder.encode(a[7], x.parms_id(), a7_scale, a7);  // L-1 (x was modswitched)
  evaluator->multiply_plain(x, a7, a7x);
  evaluator->rescale_to_next_inplace(a7x);                // L-2
  evaluator->mod_switch_to_inplace(a7x, T2.parms_id());  // L-3

  double mid_scale = (T2.scale() + a7x.scale()) / 2;
  T2.scale() = a7x.scale() = mid_scale;  // this is the correct scale now, need to set it still to avoid SEAL assert
  evaluator->add_inplace(T2, a7x);        // L-3
  evaluator->multiply_inplace(T2, x6);
  evaluator->relinearize_inplace(T2, relin_keys);
  evaluator->rescale_to_next_inplace(T2);  // L-4

  // Build T3
  troy::Ciphertext T3;
  encoder.encode(a[1], x.parms_id(), p, a1);  // L-1 (x was modswitched)
  evaluator->multiply_plain(x, a1, T3);
  evaluator->rescale_to_next_inplace(T3);  // L-2

  // T1, T2 and T3 should be on the same scale up to floating point
  // but we still need to set them manually to avoid SEAL assert
  double mid3_scale = (T1.scale() + T2.scale() + T3.scale()) / 3;
  T1.scale() = T2.scale() = T3.scale() = mid3_scale;

  dest = T2;
  evaluator->mod_switch_to_inplace(T1, dest.parms_id());  // L-4
  evaluator->add_inplace(dest, T1);
  evaluator->mod_switch_to_inplace(T3, dest.parms_id());  // L-4
  evaluator->add_inplace(dest, T3);

  /////////////////////////////////////////
  // it should be ==D but we don't stabilize if it's not, D' != D is ok
  // the goal was to make T1+T2+T3 work with minimal loss in precision
  // time_end = high_resolution_clock::now();
  // cout << "Poly eval took " << duration_cast<milliseconds>(time_end - time_start).count() << " ms" << endl;
}

troy::Ciphertext CKKSEvaluator::sgn_eval(troy::Ciphertext x, int d_g, int d_f, double sgn_factor) {
  // Compute sign function coefficients
  vector<double> f4_coeffs = F4_COEFFS;
  vector<double> g4_coeffs = G4_COEFFS;
  vector<double> f4_coeffs_last(10, 0.0);
  vector<double> g4_coeffs_last(10, 0.0);

  for (int i = 0; i <= 9; i++) {
    f4_coeffs[i] /= F4_SCALE;
    f4_coeffs_last[i] = f4_coeffs[i] * sgn_factor;

    g4_coeffs[i] /= G4_SCALE;
    g4_coeffs_last[i] = g4_coeffs[i] * sgn_factor;
  }

  troy::Ciphertext dest = x;

  for (int i = 0; i < d_g; i++) {
    if (i == d_g - 1) {
      eval_odd_deg9_poly(g4_coeffs_last, dest, dest);
    } else {
      eval_odd_deg9_poly(g4_coeffs, dest, dest);
    }
  }

  for (int i = 0; i < d_f; i++) {
    if (i == d_f - 1) {
      eval_odd_deg9_poly(f4_coeffs_last, dest, dest);
    } else {
      eval_odd_deg9_poly(f4_coeffs, dest, dest);
    }
  }

  return dest;
}

double CKKSEvaluator::calculate_MAE(vector<double> &y_true, troy::Ciphertext &ct, int N) {
  troy::Plaintext temp;
  vector<double> y_pred;

  decryptor->decrypt(ct, temp);
  encoder.decode(temp, y_pred);

  double sum_absolute_errors = 0.0;
  for (size_t i = 0; i < N; ++i) {
    sum_absolute_errors += abs(y_true[i] - y_pred[i]);
  }

  return sum_absolute_errors / N;
}

troy::Ciphertext CKKSEvaluator::exp(troy::Ciphertext x) {
  troy::Plaintext one, inverse_128;

  encoder.encode(0.0078125, x.parms_id(), x.scale(), inverse_128);
  evaluator->multiply_plain_inplace(x, inverse_128);
  evaluator->rescale_to_next_inplace(x);

  encoder.encode(1.0, x.parms_id(), x.scale(), one);
  evaluator->add_plain_inplace(x, one);

  // x^128
  for (int i = 0; i < log2(128); i++) {
    evaluator->square(x, x);
    evaluator->relinearize_inplace(x, relin_keys);
    evaluator->rescale_to_next_inplace(x);
  }

  return x;
}

troy::Ciphertext CKKSEvaluator::inverse(troy::Ciphertext x, int iter) {
  troy::Ciphertext y, tmp, res;
  troy::Plaintext one;

  encoder.encode(1.0, x.parms_id(), x.scale(), one);
  evaluator->sub_plain(x, one, y);
  evaluator->negate_inplace(y);
  evaluator->add_plain(y, one, tmp);

  res = tmp;

  for (int i = 0; i < iter; i++) {
    evaluator->square_inplace(y);
    evaluator->relinearize_inplace(y, relin_keys);
    evaluator->rescale_to_next_inplace(y);

    encoder.encode(1.0, y.parms_id(), y.scale(), one);
    evaluator->add_plain(y, one, tmp);

    evaluator->mod_switch_to_inplace(res, tmp.parms_id());
    evaluator->multiply_inplace(res, tmp);
    evaluator->relinearize_inplace(res, relin_keys);
    evaluator->rescale_to_next_inplace(res);
  }

  return res;
}


void Evaluator::add_inplace_reduced_error(troy::Ciphertext &encrypted1, const troy::Ciphertext &encrypted2) {
  size_t encrypted1_coeff_modulus_size = encrypted1.coeff_modulus_size();
  size_t encrypted2_coeff_modulus_size = encrypted2.coeff_modulus_size();

  if (encrypted1_coeff_modulus_size == encrypted2_coeff_modulus_size) {
    encrypted1.scale() = encrypted2.scale();
    add_inplace(encrypted1, encrypted2);
    return;
  }

  else if (encrypted1_coeff_modulus_size < encrypted2_coeff_modulus_size) {
    auto &context_data = *context->get_context_data(encrypted2.parms_id())->get();
    auto &parms = context_data.parms();
    auto modulus = parms.coeff_modulus();
    troy::Ciphertext encrypted2_adjusted;

    double scale_adjust = encrypted1.scale() * static_cast<double>(modulus[encrypted2_coeff_modulus_size - 1].value()) / (encrypted2.scale() * encrypted2.scale());
    multiply_const(encrypted2, scale_adjust, encrypted2_adjusted);
    encrypted2_adjusted.scale() = encrypted1.scale() * static_cast<double>(modulus[encrypted2_coeff_modulus_size - 1].value());
    rescale_to_next_inplace(encrypted2_adjusted);
    mod_switch_to_inplace(encrypted2_adjusted, encrypted1.parms_id());
    encrypted1.scale() = encrypted2_adjusted.scale();
    add_inplace(encrypted1, encrypted2_adjusted);
  }

  else {
    auto &context_data = *context->get_context_data(encrypted1.parms_id())->get();
    auto &parms = context_data.parms();
    auto modulus = parms.coeff_modulus();
    troy::Ciphertext encrypted1_adjusted;

    double scale_adjust = encrypted2.scale() * static_cast<double>(modulus[encrypted1_coeff_modulus_size - 1].value()) / (encrypted1.scale() * encrypted1.scale());
    multiply_const(encrypted1, scale_adjust, encrypted1_adjusted);
    encrypted1_adjusted.scale() = encrypted2.scale() * static_cast<double>(modulus[encrypted1_coeff_modulus_size - 1].value());
    rescale_to_next_inplace(encrypted1_adjusted);
    mod_switch_to_inplace(encrypted1_adjusted, encrypted2.parms_id());
    encrypted1_adjusted.scale() = encrypted2.scale();
    add(encrypted1_adjusted, encrypted2, encrypted1);
  }
}

void Evaluator::sub_inplace_reduced_error(troy::Ciphertext &encrypted1, const troy::Ciphertext &encrypted2) {
  size_t encrypted1_coeff_modulus_size = encrypted1.coeff_modulus_size();
  size_t encrypted2_coeff_modulus_size = encrypted2.coeff_modulus_size();

  if (encrypted1_coeff_modulus_size == encrypted2_coeff_modulus_size) {
    encrypted1.scale() = encrypted2.scale();
    sub_inplace(encrypted1, encrypted2);
    return;
  }

  else if (encrypted1_coeff_modulus_size < encrypted2_coeff_modulus_size) {
    auto &context_data = *context->get_context_data(encrypted2.parms_id())->get();
    auto &parms = context_data.parms();
    auto modulus = parms.coeff_modulus();
    troy::Ciphertext encrypted2_adjusted;

    double scale_adjust = encrypted1.scale() * static_cast<double>(modulus[encrypted2_coeff_modulus_size - 1].value()) / (encrypted2.scale() * encrypted2.scale());
    multiply_const(encrypted2, scale_adjust, encrypted2_adjusted);
    encrypted2_adjusted.scale() = encrypted1.scale() * static_cast<double>(modulus[encrypted2_coeff_modulus_size - 1].value());
    rescale_to_next_inplace(encrypted2_adjusted);
    mod_switch_to_inplace(encrypted2_adjusted, encrypted1.parms_id());
    encrypted1.scale() = encrypted2_adjusted.scale();
    sub_inplace(encrypted1, encrypted2_adjusted);
  }

  else {
    auto &context_data = *context->get_context_data(encrypted1.parms_id())->get();
    auto &parms = context_data.parms();
    auto modulus = parms.coeff_modulus();
    troy::Ciphertext encrypted1_adjusted;

    double scale_adjust = encrypted2.scale() * static_cast<double>(modulus[encrypted1_coeff_modulus_size - 1].value()) / (encrypted1.scale() * encrypted1.scale());
    multiply_const(encrypted1, scale_adjust, encrypted1_adjusted);
    encrypted1_adjusted.scale() = encrypted2.scale() * static_cast<double>(modulus[encrypted1_coeff_modulus_size - 1].value());
    rescale_to_next_inplace(encrypted1_adjusted);
    mod_switch_to_inplace(encrypted1_adjusted, encrypted2.parms_id());
    encrypted1_adjusted.scale() = encrypted2.scale();
    sub(encrypted1_adjusted, encrypted2, encrypted1);
  }
}

void Evaluator::multiply_inplace_reduced_error(troy::Ciphertext &encrypted1, const troy::Ciphertext &encrypted2, const troy::RelinKeys &relin_keys) {
  size_t encrypted1_coeff_modulus_size = encrypted1.coeff_modulus_size();
  size_t encrypted2_coeff_modulus_size = encrypted2.coeff_modulus_size();

  if (encrypted1_coeff_modulus_size == encrypted2_coeff_modulus_size) {
    encrypted1.scale() = encrypted2.scale();
    multiply_inplace(encrypted1, encrypted2);
    relinearize_inplace(encrypted1, relin_keys);
    return;
  }

  else if (encrypted1_coeff_modulus_size < encrypted2_coeff_modulus_size) {
    auto &context_data = *context->get_context_data(encrypted2.parms_id())->get();
    auto &parms = context_data.parms();
    auto modulus = parms.coeff_modulus();
    troy::Ciphertext encrypted2_adjusted;

    double scale_adjust = encrypted1.scale() * static_cast<double>(modulus[encrypted2_coeff_modulus_size - 1].value()) / (encrypted2.scale() * encrypted2.scale());
    multiply_const(encrypted2, scale_adjust, encrypted2_adjusted);
    encrypted2_adjusted.scale() = encrypted1.scale() * static_cast<double>(modulus[encrypted2_coeff_modulus_size - 1].value());
    rescale_to_next_inplace(encrypted2_adjusted);
    mod_switch_to_inplace(encrypted2_adjusted, encrypted1.parms_id());
    encrypted1.scale() = encrypted2_adjusted.scale();
    multiply_inplace(encrypted1, encrypted2_adjusted);
  }

  else {
    auto &context_data = *context->get_context_data(encrypted1.parms_id())->get();
    auto &parms = context_data.parms();
    auto modulus = parms.coeff_modulus();
    troy::Ciphertext encrypted1_adjusted;

    double scale_adjust = encrypted2.scale() * static_cast<double>(modulus[encrypted1_coeff_modulus_size - 1].value()) / (encrypted1.scale() * encrypted1.scale());
    multiply_const(encrypted1, scale_adjust, encrypted1_adjusted);
    encrypted1_adjusted.scale() = encrypted2.scale() * static_cast<double>(modulus[encrypted1_coeff_modulus_size - 1].value());
    rescale_to_next_inplace(encrypted1_adjusted);
    mod_switch_to_inplace(encrypted1_adjusted, encrypted2.parms_id());
    encrypted1_adjusted.scale() = encrypted2.scale();
    multiply(encrypted1_adjusted, encrypted2, encrypted1);
  }

  relinearize_inplace(encrypted1, relin_keys);
}

};